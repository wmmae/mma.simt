#include "hip/hip_runtime.h"
#include <iostream>
#include <random>
#include <wmma_extension/mma_simt.hpp>
#include "utils.hpp"

namespace {
template <class T>
constexpr double error_threshold = 0.0;
template <>
constexpr double error_threshold<half > = 1e-3;
template <>
constexpr double error_threshold<float> = 1e-5;
} // noname namespace

template <unsigned N, class T, class A_Layout, class B_Layout>
__global__ void mma_kernel_abcd(float* const d_ptr, const float* const a_ptr, const float* const b_ptr, const float* const c_ptr, const nvcuda::wmma::layout_t cd_layout) {
	constexpr unsigned LD = N;
	__shared__ float smem[N * LD];
	mtk::test_utils::fill_zero(smem, N * LD);

	mtk::wmma::mma_simt::fragment<nvcuda::wmma::matrix_a   , N, N, N, T, A_Layout> frag_a;
	mtk::wmma::mma_simt::fragment<nvcuda::wmma::matrix_b   , N, N, N, T, B_Layout> frag_b;
	mtk::wmma::mma_simt::fragment<nvcuda::wmma::accumulator, N, N, N, T, void    > frag_c, frag_d;
	// Load A
	mtk::test_utils::copy_matrix(smem, LD, a_ptr, N, N, N);
	mtk::wmma::mma_simt::load_matrix_sync(frag_a, smem, LD);

	// Load B
	mtk::test_utils::copy_matrix(smem, LD, b_ptr, N, N, N);
	mtk::wmma::mma_simt::load_matrix_sync(frag_b, smem, LD);

	// Load C
	mtk::test_utils::copy_matrix(smem, LD, c_ptr, N, N, N);
	mtk::wmma::mma_simt::load_matrix_sync(frag_c, smem, LD, cd_layout);

	// Fill D
	mtk::wmma::mma_simt::fill_fragment(frag_d, 0.0f);

	// mma
	mtk::wmma::mma_simt::mma_sync(frag_d, frag_a, frag_b, frag_c);

	// Store D
	mtk::wmma::mma_simt::store_matrix_sync(smem, frag_d, LD, cd_layout);
	mtk::test_utils::copy_matrix(d_ptr, N, smem, LD, N, N);

	// Test for fill_zero
	mtk::wmma::mma_simt::fill_zero(frag_d);
}

template <unsigned N, class T, class A_Layout, class B_Layout>
void test_mma(const nvcuda::wmma::layout_t cd_layout) {
	float *hA, *hB, *hC, *hD;
	hipHostMalloc(&hA, N * N * sizeof(float));
	hipHostMalloc(&hB, N * N * sizeof(float));
	hipHostMalloc(&hC, N * N * sizeof(float));
	hipHostMalloc(&hD, N * N * sizeof(float));

	std::mt19937 mt(std::random_device{}());
	std::uniform_real_distribution<float> dist(-1.0f, 1.0f);

	for (unsigned i = 0; i < N * N; i++) {
			hA[i] = dist(mt);
			hB[i] = dist(mt);
			hC[i] = dist(mt);
	}
	hipDeviceSynchronize();

	mma_kernel_abcd<N, T, A_Layout, B_Layout><<<1, mtk::test_utils::warp_size>>>(hD, hA, hB, hC, cd_layout);

	const auto stat = hipDeviceSynchronize();
	if (stat != hipSuccess) {
		std::printf("[error] %s\n", hipGetErrorString(stat));
	}

	double max_error = 0.;
	for (unsigned m = 0; m < N; m++) {
		for (unsigned n = 0; n < N; n++) {
			double cor_d = 0.;
			for (unsigned k = 0; k < N; k++) {
				const auto a_mem_index = std::is_same<A_Layout, nvcuda::wmma::col_major>::value ? (k * N + m) : (m * N + k);
				const auto b_mem_index = std::is_same<B_Layout, nvcuda::wmma::col_major>::value ? (k + n * N) : (n + k * N);
				cor_d += static_cast<double>(hA[a_mem_index]) * static_cast<double>(hB[b_mem_index]);
			}
			const auto c_mem_index = (cd_layout == nvcuda::wmma::mem_col_major) ? (m + n * N) : (n + m * N);
			cor_d += hC[c_mem_index];

			max_error = std::max(max_error, std::abs(cor_d - hD[c_mem_index]));
		}
	}

	std::printf(
			"[Type:%5s, N:%3u, A_Layout:%10s, B_Layout:%10s, C_Layout:%10s, FragShape<%2d,%2d,%2d>] max_error: %e (%6s)\n",
			mtk::test_utils::to_string<T>().c_str(),
			N,
			mtk::test_utils::to_string<A_Layout>().c_str(),
			mtk::test_utils::to_string<B_Layout>().c_str(),
			(cd_layout == nvcuda::wmma::mem_col_major) ? mtk::test_utils::to_string<nvcuda::wmma::col_major>().c_str() : mtk::test_utils::to_string<nvcuda::wmma::row_major>().c_str(),
			N, N, N,
			max_error,
			(max_error < error_threshold<T> ? "PASSED" : "FAILED")
			);

	hipHostFree(hA);
	hipHostFree(hB);
	hipHostFree(hC);
	hipHostFree(hD);
}

int main() {
	// wmma FP16 test
	test_mma<16, half , nvcuda::wmma::col_major, nvcuda::wmma::col_major>(nvcuda::wmma::mem_col_major);
	test_mma<16, half , nvcuda::wmma::col_major, nvcuda::wmma::col_major>(nvcuda::wmma::mem_col_major);
	test_mma<16, half , nvcuda::wmma::row_major, nvcuda::wmma::col_major>(nvcuda::wmma::mem_col_major);
	test_mma<16, half , nvcuda::wmma::row_major, nvcuda::wmma::col_major>(nvcuda::wmma::mem_col_major);
	test_mma<16, half , nvcuda::wmma::col_major, nvcuda::wmma::row_major>(nvcuda::wmma::mem_col_major);
	test_mma<16, half , nvcuda::wmma::col_major, nvcuda::wmma::row_major>(nvcuda::wmma::mem_col_major);
	test_mma<16, half , nvcuda::wmma::row_major, nvcuda::wmma::row_major>(nvcuda::wmma::mem_col_major);
	test_mma<16, half , nvcuda::wmma::row_major, nvcuda::wmma::row_major>(nvcuda::wmma::mem_col_major);
	test_mma<16, half , nvcuda::wmma::col_major, nvcuda::wmma::col_major>(nvcuda::wmma::mem_row_major);
	test_mma<16, half , nvcuda::wmma::col_major, nvcuda::wmma::col_major>(nvcuda::wmma::mem_row_major);
	test_mma<16, half , nvcuda::wmma::row_major, nvcuda::wmma::col_major>(nvcuda::wmma::mem_row_major);
	test_mma<16, half , nvcuda::wmma::row_major, nvcuda::wmma::col_major>(nvcuda::wmma::mem_row_major);
	test_mma<16, half , nvcuda::wmma::col_major, nvcuda::wmma::row_major>(nvcuda::wmma::mem_row_major);
	test_mma<16, half , nvcuda::wmma::col_major, nvcuda::wmma::row_major>(nvcuda::wmma::mem_row_major);
	test_mma<16, half , nvcuda::wmma::row_major, nvcuda::wmma::row_major>(nvcuda::wmma::mem_row_major);
	test_mma<16, half , nvcuda::wmma::row_major, nvcuda::wmma::row_major>(nvcuda::wmma::mem_row_major);
	test_mma<16, half , nvcuda::wmma::col_major, nvcuda::wmma::col_major>(nvcuda::wmma::mem_col_major);
	test_mma<16, half , nvcuda::wmma::col_major, nvcuda::wmma::col_major>(nvcuda::wmma::mem_col_major);
	test_mma<16, float, nvcuda::wmma::row_major, nvcuda::wmma::col_major>(nvcuda::wmma::mem_col_major);
	test_mma<16, float, nvcuda::wmma::row_major, nvcuda::wmma::col_major>(nvcuda::wmma::mem_col_major);
	test_mma<16, float, nvcuda::wmma::col_major, nvcuda::wmma::row_major>(nvcuda::wmma::mem_col_major);
	test_mma<16, float, nvcuda::wmma::col_major, nvcuda::wmma::row_major>(nvcuda::wmma::mem_col_major);
	test_mma<16, float, nvcuda::wmma::row_major, nvcuda::wmma::row_major>(nvcuda::wmma::mem_col_major);
	test_mma<16, float, nvcuda::wmma::row_major, nvcuda::wmma::row_major>(nvcuda::wmma::mem_col_major);
	test_mma<16, float, nvcuda::wmma::col_major, nvcuda::wmma::col_major>(nvcuda::wmma::mem_row_major);
	test_mma<16, float, nvcuda::wmma::col_major, nvcuda::wmma::col_major>(nvcuda::wmma::mem_row_major);
	test_mma<16, float, nvcuda::wmma::row_major, nvcuda::wmma::col_major>(nvcuda::wmma::mem_row_major);
	test_mma<16, float, nvcuda::wmma::row_major, nvcuda::wmma::col_major>(nvcuda::wmma::mem_row_major);
	test_mma<16, float, nvcuda::wmma::col_major, nvcuda::wmma::row_major>(nvcuda::wmma::mem_row_major);
	test_mma<16, float, nvcuda::wmma::col_major, nvcuda::wmma::row_major>(nvcuda::wmma::mem_row_major);
	test_mma<16, float, nvcuda::wmma::row_major, nvcuda::wmma::row_major>(nvcuda::wmma::mem_row_major);
	test_mma<16, float, nvcuda::wmma::row_major, nvcuda::wmma::row_major>(nvcuda::wmma::mem_row_major);
}
